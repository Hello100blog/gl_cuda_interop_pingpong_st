#include "hip/hip_runtime.h"

__constant__ unsigned int  volumeDim;
__constant__ float  cellWidth;
__constant__ float  volumeCoeff;
__constant__ float  alpha;

surface<void, 3> volumeTexOut;
texture<float, 3, hipReadModeElementType> volumeTexIn;

extern "C"  __global__ void heatEquationStep(float dt)
{
	int i = 1 + blockIdx.x * blockDim.x + threadIdx.x ;
	int j = 1 + blockIdx.y * blockDim.y + threadIdx.y ;
	int k = 1 + blockIdx.z * blockDim.z + threadIdx.z ;
	
	float val = tex3D(  volumeTexIn, i,     j,   k   );
	float valU = tex3D( volumeTexIn, (i+1), j,   k   );
	float valD = tex3D( volumeTexIn, (i-1), j,   k   );
	float valN = tex3D( volumeTexIn, i,     j+1, k   );
	float valS = tex3D( volumeTexIn, i,     j-1, k   );
	float valE = tex3D( volumeTexIn, i,     j,   k+1 );
	float valW = tex3D( volumeTexIn, i,     j,   k-1 );
	
	float negSixVal = -6.0f*val;

	float result = val + dt * alpha * (negSixVal + valU + valD + valN + valS + valE + valW) / (2.0f * cellWidth * cellWidth);

    surf3Dwrite(result, volumeTexOut, i * sizeof(float), j, k);
}